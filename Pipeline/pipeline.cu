#include "hip/hip_runtime.h"
// -*- c++ -*-
/***************************************************************************
 *
 *   Copyright (C) 2012 by Ben Barsdell and Andrew Jameson
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/

#include <vector>
#include <memory>
#include <iostream>
using std::cout;
using std::cerr;
using std::endl;
#include <sstream>
#include <iomanip>
#include <string>
#include <fstream>
#include <time.h>
#include <algorithm>
#include <functional>

#include <src/sigproc.h>
#include <src/header.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
using thrust::host_vector;
using thrust::device_vector;
#include <thrust/version.h>
#include <thrust/copy.h>
#include <thrust/reduce.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/gather.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/fill.h>
#include <thrust/scan.h>
#include <thrust/extrema.h>
#include <thrust/functional.h>

#include "hd/pipeline.h"
#include "hd/maths.h"
#include "hd/clean_filterbank_rfi.h"
#include "hd/strided_range.h"

#include "hd/remove_baseline.h"
#include "hd/matched_filter.h"
#include "hd/get_rms.h"
#include "hd/find_giants.h"
#include "hd/label_candidate_clusters.h"
#include "hd/merge_candidates.h"

#include "hd/DataSource.h"
#include "hd/ClientSocket.h"
#include "hd/SocketException.h"
#include "hd/stopwatch.h"         // For benchmarking

#include <dedisp.h>

FILE *output;

void tfunc(std::vector<hd_byte> &vec) {

  hd_byte* ddata = (hd_byte *)malloc(sizeof(hd_byte)*200);
  std::copy(vec.begin(),vec.end(),ddata);
  std::copy(vec.begin(),vec.end(),ddata+100);
    
  for (int i=0;i<200;i++)
    cout << +ddata[i] << " ";
  cout << " " << endl;

  free(ddata);
  
}

void send_string(char *string) /* includefile */
{
  int len;
  len=strlen(string);
  fwrite(&len, sizeof(int), 1, output);
  fwrite(string, sizeof(char), len, output);
}

void send_float(char *name,float floating_point) /* includefile */
{
  send_string(name);
  fwrite(&floating_point,sizeof(float),1,output);
}

void send_double (char *name, double double_precision) /* includefile */
{
  send_string(name);
  fwrite(&double_precision,sizeof(double),1,output);
}

void send_int(char *name, int integer) /* includefile */
{
  send_string(name);
  fwrite(&integer,sizeof(int),1,output);
}

void send_char(char *name, char integer) /* includefile */
{
  send_string(name);
  fwrite(&integer,sizeof(char),1,output);
}


void send_long(char *name, long integer) /* includefile */
{
  send_string(name);
  fwrite(&integer,sizeof(long),1,output);
}

void send_coords(double raj, double dej, double az, double za) /*includefile*/
{
  if ((raj != 0.0) || (raj != -1.0)) send_double("src_raj",raj);
  if ((dej != 0.0) || (dej != -1.0)) send_double("src_dej",dej);
  if ((az != 0.0)  || (az != -1.0))  send_double("az_start",az);
  if ((za != 0.0)  || (za != -1.0))  send_double("za_start",za);
}

#define HD_BENCHMARK

#ifdef HD_BENCHMARK
  void start_timer(Stopwatch& timer) { timer.start(); }
  void stop_timer(Stopwatch& timer) { hipDeviceSynchronize(); timer.stop(); }
#else
  void start_timer(Stopwatch& timer) { }
  void stop_timer(Stopwatch& timer) { }
#endif // HD_BENCHMARK

#include <utility> // For std::pair
template<typename T, typename U>
std::pair<T&,U&> tie(T& a, U& b) { return std::pair<T&,U&>(a,b); }

struct hd_pipeline_t {
  hd_params   params;
  dedisp_plan dedispersion_plan;
  //MPI_Comm    communicator;

  // Memory buffers used during pipeline execution
  std::vector<hd_byte>    h_clean_filterbank;
  host_vector<hd_byte>    h_dm_series;
  device_vector<hd_float> d_time_series;
  device_vector<hd_float> d_filtered_series;
};

hd_error allocate_gpu(const hd_pipeline pl) {
  // TODO: This is just a simple proc-->GPU heuristic to get us started
  int gpu_count;
  hipGetDeviceCount(&gpu_count);
  //int proc_idx;
  //MPI_Comm comm = pl->communicator;
  //MPI_Comm_rank(comm, &proc_idx);
  int proc_idx = pl->params.beam;
  int gpu_idx = pl->params.gpu_id;
  
  hipError_t cerror = hipSetDevice(gpu_idx);
  if( cerror != hipSuccess ) {
    cerr << "Could not setCudaDevice to " << gpu_idx << ": " << hipGetErrorString(cerror) <<  endl;
    return throw_cuda_error(cerror);
  }
  
  if( pl->params.verbosity >= 1 ) {
    cout << "Process " << proc_idx << " using GPU " << gpu_idx << endl;
  }
  
  if( !pl->params.yield_cpu ) {
    if( pl->params.verbosity >= 2 ) {
      cout << "\tProcess " << proc_idx << " setting CPU to spin" << endl;
    }
    cerror = hipSetDeviceFlags(hipDeviceScheduleSpin);
    if( cerror != hipSuccess ) {
      return throw_cuda_error(cerror);
    }
  }
  else {
    if( pl->params.verbosity >= 2 ) {
      cout << "\tProcess " << proc_idx << " setting CPU to yield" << endl;
    }
    // Note: This Yield flag doesn't seem to work properly.
    //   The BlockingSync flag does the job, although it may interfere
    //     with GPU/CPU overlapping (not currently used).
    //cerror = hipSetDeviceFlags(hipDeviceScheduleYield);
    cerror = hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
    if( cerror != hipSuccess ) {
      return throw_cuda_error(cerror);
    }
  }
  
  return HD_NO_ERROR;
}


unsigned int get_filter_index(unsigned int filter_width) {
  // This function finds log2 of the 32-bit power-of-two number v
  unsigned int v = filter_width;
  static const unsigned int b[] = {0xAAAAAAAA, 0xCCCCCCCC, 0xF0F0F0F0, 
                                   0xFF00FF00, 0xFFFF0000};
  register unsigned int r = (v & b[0]) != 0;
  for( int i=4; i>0; --i) {
    r |= ((v & b[i]) != 0) << i;
  }
  return r;
}


hd_error hd_create_pipeline(hd_pipeline* pipeline_, hd_params params) {
  *pipeline_ = 0;
  
  // Note: We use a smart pointer here to automatically clean up after errors
  typedef std::auto_ptr<hd_pipeline_t> smart_pipeline_ptr;
  smart_pipeline_ptr pipeline = smart_pipeline_ptr(new hd_pipeline_t());
  if( !pipeline.get() ) {
    return throw_error(HD_MEM_ALLOC_FAILED);
  }
  
  pipeline->params = params;
  
  if( params.verbosity >= 2 ) {
    cout << "\tAllocating GPU..." << endl;
  }
  
  hd_error error = allocate_gpu(pipeline.get());
  if( error != HD_NO_ERROR ) {
    return throw_error(error);
  }
  
  if( params.verbosity >= 1 ) {
    cout << "nchans = " << params.nchans << endl;
    cout << "dt     = " << params.dt << endl;
    cout << "f0     = " << params.f0 << endl;
    cout << "df     = " << params.df << endl;
    cout << "nsnap     = " << params.nsnap << endl;
  }
  
  if( params.verbosity >= 2 ) {
    cout << "\tCreating dedispersion plan..." << endl;
  }
  
  dedisp_error derror;
  derror = dedisp_create_plan(&pipeline->dedispersion_plan,
                              params.nchans, params.dt,
                              params.f0, params.df);
  if( derror != DEDISP_NO_ERROR ) {
    return throw_dedisp_error(derror);
  }
  // TODO: Consider loading a pre-generated DM list instead for flexibility
  derror = dedisp_generate_dm_list(pipeline->dedispersion_plan,
                                   pipeline->params.dm_min,
                                   pipeline->params.dm_max,
                                   pipeline->params.dm_pulse_width,
                                   pipeline->params.dm_tol);
  if( derror != DEDISP_NO_ERROR ) {
    return throw_dedisp_error(derror);
  }
  
  if( pipeline->params.use_scrunching ) {
    derror = dedisp_enable_adaptive_dt(pipeline->dedispersion_plan,
                                       pipeline->params.dm_pulse_width,
                                       pipeline->params.scrunch_tol);
    if( derror != DEDISP_NO_ERROR ) {
      return throw_dedisp_error(derror);
    }
  }
  
  *pipeline_ = pipeline.release();
  
  if( params.verbosity >= 2 ) {
    cout << "\tInitialisation complete." << endl;
  }
  
  if( params.verbosity >= 1 ) {
    cout << "Using Thrust v"
         << THRUST_MAJOR_VERSION << "."
         << THRUST_MINOR_VERSION << "."
         << THRUST_SUBMINOR_VERSION << endl;
  }
  
  return HD_NO_ERROR;
}

hd_error hd_execute(hd_pipeline pl,
                    const hd_byte* h_filterbank, hd_size nsamps, hd_size nbits,
                    hd_size first_idx, hd_size iidx, hd_size* nsamps_processed, hd_size gulp_idx) {
  hd_error error = HD_NO_ERROR;

  cout << "working on gulp " << gulp_idx << endl;
  
  Stopwatch total_timer;
  Stopwatch memory_timer;
  Stopwatch clean_timer;
  Stopwatch dedisp_timer;
  Stopwatch communicate_timer;
  Stopwatch copy_timer;
  Stopwatch baseline_timer;
  Stopwatch normalise_timer;
  Stopwatch filter_timer;
  Stopwatch coinc_timer;
  Stopwatch giants_timer;
  Stopwatch candidates_timer;
  
  start_timer(total_timer);

  start_timer(clean_timer);
  // Note: Filterbank cleaning must be done out-of-place
  hd_size nbytes = (nsamps) * pl->params.nchans * nbits / 8 * pl->params.nbeams;
  start_timer(memory_timer);
  pl->h_clean_filterbank.resize(nbytes);
  std::vector<int>          h_killmask(pl->params.nchans, 1);
  stop_timer(memory_timer);

  // copy to clean filterbank
  std::copy(h_filterbank,h_filterbank+nbytes,pl->h_clean_filterbank.begin());
  
  // apply manual killmasks
  error = apply_manual_killmasks (pl->dedispersion_plan,
                                  &h_killmask[0], 
                                  pl->params.num_channel_zaps,
                                  pl->params.channel_zaps);
  if( error != HD_NO_ERROR ) {
    return throw_error(error);
  }
  
  hd_size good_chan_count = thrust::reduce(h_killmask.begin(),
                                           h_killmask.end());
  hd_size bad_chan_count = pl->params.nchans - good_chan_count;
  if( pl->params.verbosity >= 2 ) {
    cout << "Bad channel count = " << bad_chan_count << endl;
  }
  
  stop_timer(clean_timer);
  
  if( pl->params.verbosity >= 2 ) {
    cout << "\tGenerating DM list..." << endl;
  }
  
  if( pl->params.verbosity >= 3 ) {
    cout << "dm_min = " << pl->params.dm_min << endl;
    cout << "dm_max = " << pl->params.dm_max << endl;
    cout << "dm_tol = " << pl->params.dm_tol << endl;
    cout << "dm_pulse_width = " << pl->params.dm_pulse_width << endl;
    cout << "nchans = " << pl->params.nchans << endl;
    cout << "dt = " << pl->params.dt << endl;
    
    cout << "dedisp nchans = " << dedisp_get_channel_count(pl->dedispersion_plan) << endl;
    cout << "dedisp dt = " << dedisp_get_dt(pl->dedispersion_plan) << endl;
    cout << "dedisp f0 = " << dedisp_get_f0(pl->dedispersion_plan) << endl;
    cout << "dedisp df = " << dedisp_get_df(pl->dedispersion_plan) << endl;
  }
  
  hd_size      dm_count = dedisp_get_dm_count(pl->dedispersion_plan);
  const float* dm_list  = dedisp_get_dm_list(pl->dedispersion_plan);
  
  const dedisp_size* scrunch_factors =
    dedisp_get_dt_factors(pl->dedispersion_plan);
  if (pl->params.verbosity >= 3 ) 
  {
    cout << "DM List for " << pl->params.dm_min << " to " << pl->params.dm_max << endl;
    for( hd_size i=0; i<dm_count; ++i ) {
      cout << dm_list[i] << endl;
    }
  }  

  if( pl->params.verbosity >= 2 ) {
    cout << "Scrunch factors:" << endl;
    for( hd_size i=0; i<dm_count; ++i ) {
      cout << scrunch_factors[i] << " ";
    }
    cout << endl;
  }
  
  // Set channel killmask for dedispersion
  dedisp_set_killmask(pl->dedispersion_plan, &h_killmask[0]);
  hd_size nsamps_computed  = nsamps - dedisp_get_max_delay(pl->dedispersion_plan);
  hd_size series_stride    = nsamps_computed;
  
  // Report the number of samples that will be properly processed
  *nsamps_processed = nsamps_computed - pl->params.boxcar_max;
  
  if( pl->params.verbosity >= 3 ) {
    cout << "dm_count = " << dm_count << endl;
    cout << "max delay = " << dedisp_get_max_delay(pl->dedispersion_plan) << endl;
    cout << "nsamps_computed = " << nsamps_computed << endl;
  }
  
  hd_size beam = pl->params.beam;
  
  if( pl->params.verbosity >= 2 ) {
    cout << "\tAllocating memory for pipeline computations..." << endl;
  }
  
  start_timer(memory_timer);
  
  pl->h_dm_series.resize((nsamps *( pl->params.nbeams -1 ) + series_stride) * pl->params.dm_nbits/8 * dm_count);
  pl->d_time_series.resize(series_stride+(pl->params.nbeams-1)*nsamps);
  pl->d_filtered_series.resize(series_stride+(pl->params.nbeams-1)*nsamps, 0);
  
  stop_timer(memory_timer);

  GetRMSPlan rms_getter;
  RemoveBaselinePlan          baseline_remover;
  MatchedFilterPlan<hd_float> matched_filter_plan;
  GiantFinder                 giant_finder;
  
  thrust::device_vector<hd_float> d_giant_peaks;
  thrust::device_vector<hd_size>  d_giant_inds;
  thrust::device_vector<hd_size>  d_giant_begins;
  thrust::device_vector<hd_size>  d_giant_ends;
  thrust::device_vector<hd_size>  d_giant_filter_inds;
  thrust::device_vector<hd_size>  d_giant_dm_inds;
  thrust::device_vector<hd_size>  d_giant_members;
  
  typedef thrust::device_ptr<hd_float> dev_float_ptr;
  typedef thrust::device_ptr<hd_size>  dev_size_ptr;
  
  if( pl->params.verbosity >= 2 ) {
    cout << "\tDedispersing for DMs " << dm_list[0]
         << " to " << dm_list[dm_count-1] << "..." << endl;
  }
  
  // Dedisperse
  dedisp_error       derror;
  const dedisp_byte* in = &pl->h_clean_filterbank[0];
  dedisp_byte*       out = &pl->h_dm_series[0];
  dedisp_size        in_nbits = nbits;
  dedisp_size        in_stride = pl->params.nchans * in_nbits/8;
  dedisp_size        out_nbits = pl->params.dm_nbits;
  dedisp_size        out_stride = series_stride * out_nbits/8  + (pl->params.nbeams - 1) * nsamps * out_nbits/8;
  unsigned           flags = 0;
  start_timer(dedisp_timer);
  derror = dedisp_execute_adv(pl->dedispersion_plan, nsamps*pl->params.nbeams,
                              in, in_nbits, in_stride,
                              out, out_nbits, out_stride,
                              flags);
/*FILE *dedisp_out;
   char ofiledo[200];
   sprintf(ofiledo,"%s/dedisp_out.cand",pl->params.output_dir);
dedisp_out = fopen(ofiledo,"a");
*hd_float* dummy;
int* dummy2;
for (int i=0; i < pl->h_dm_series.size()/4;i++)  {
dummy = (hd_float*)&pl->h_dm_series[i*4];
dummy2 = (int*)&pl->h_dm_series[i*4];
cout << "int " << *dummy2 << endl;
cout << *dummy << endl;
fprintf(dedisp_out,"%g\n",*dummy);
}*/

  //remove beam parts with overlap or keep them and remove giants in overlap region later
  stop_timer(dedisp_timer);
  /*if (pl->params.nbeams > 1) {
  for(hd_size beam = 0; beam < pl->params.nbeams; beam++)  {
    for(hd_size dm_trial = 0; dm_trial < dm_count; dm_trial++)  {
      std::copy(&pl->h_dm_series[(beam*dm_count+dm_trial)*nsamps*out_nbits/8],
      &pl->h_dm_series[(beam*dm_count+dm_trial)*(nsamps)*out_nbits/8+ nsamps_computed*out_nbits/8],
      &pl->h_dm_series[(beam*dm_count+dm_trial)*(series_stride)*out_nbits/8]); 
    }
  }
  pl->h_dm_series.erase(&pl->h_dm_series[pl->params.nbeams*dm_count*series_stride],&pl->h_dm_series[pl->h_dm_series.size()]);
  }*/
  if( derror != DEDISP_NO_ERROR ) {
    return throw_dedisp_error(derror);
  }
    
  if( pl->params.verbosity >= 2 ) {
    cout << "\tBeginning inner pipeline..." << endl;
  }
  
  bool too_many_giants = false;
  int notrig = 0;

  // For each DM

  for( hd_size dm_idx=0; dm_idx<dm_count; ++dm_idx ) {

    hd_size  cur_dm_scrunch = scrunch_factors[dm_idx];
    hd_size  cur_nsamps  = (nsamps_computed+nsamps*(pl->params.nbeams-1)) / cur_dm_scrunch;
    hd_float cur_dt      = pl->params.dt * cur_dm_scrunch;

    // Bail if the candidate rate is too high
    if( too_many_giants ) {
      break;
    }
    
    if( pl->params.verbosity >= 4 ) {
      cout << "dm_idx     = " << dm_idx << endl;
      cout << "scrunch    = " << scrunch_factors[dm_idx] << endl;
      cout << "cur_nsamps = " << cur_nsamps << endl;
      cout << "dt0        = " << pl->params.dt << endl;
      cout << "cur_dt     = " << cur_dt << endl;
        
      cout << "\tBaselining and normalising each beam..." << endl;
    }
    
    hd_float* time_series = thrust::raw_pointer_cast(&pl->d_time_series[0]);
    
    // Copy the time series to the device and convert to floats
    hd_size offset = dm_idx * (series_stride + (pl->params.nbeams-1)*nsamps) * pl->params.dm_nbits/8;
    start_timer(copy_timer);
    switch( pl->params.dm_nbits ) {
    case 8:
      thrust::copy((unsigned char*)&pl->h_dm_series[offset],
                   (unsigned char*)&pl->h_dm_series[offset] + cur_nsamps,
                   pl->d_time_series.begin());
      break;
    case 16:
      thrust::copy((unsigned short*)&pl->h_dm_series[offset],
                   (unsigned short*)&pl->h_dm_series[offset] + cur_nsamps,
                   pl->d_time_series.begin());
      break;
    case 32:
      // Note: 32-bit implies float, not unsigned int
      thrust::copy((float*)&pl->h_dm_series[offset],
                   (float*)&pl->h_dm_series[offset] + cur_nsamps,
                   pl->d_time_series.begin());
      break;
    default:
      return HD_INVALID_NBITS;
    }
/*FILE *dm_out;
   char ofiledmo[200];
   sprintf(ofiledmo,"%s/dm_out.cand",pl->params.output_dir);
   dm_out = fopen(ofiledmo,"a");
for (int l=0; l < cur_nsamps;l++)  {
fprintf(dm_out,"dm_idx %d",dm_idx);
fprintf(dm_out,"%g\n",pl->h_dm_series[offset*8/pl->params.dm_nbits+l]);
}*/
    stop_timer(copy_timer);
    
    // Remove the baseline
    // -------------------
    // Note: Divided by 2 to form a smoothing radius
    hd_size nsamps_smooth = hd_size(pl->params.baseline_length / (2 * cur_dt));
    // Crop the smoothing length in case not enough samples
    start_timer(baseline_timer);
    
    // TESTING
    error = baseline_remover.exec(time_series, cur_nsamps, nsamps_smooth);
    stop_timer(baseline_timer);
    if( error != HD_NO_ERROR ) {
      return throw_error(error);
    }
      
    // Normalise
    // ---------
    start_timer(normalise_timer);
    hd_float rms = rms_getter.exec(time_series, cur_nsamps);
    thrust::transform(pl->d_time_series.begin(), pl->d_time_series.end(),
                      thrust::make_constant_iterator(hd_float(1.0)/rms),
                      pl->d_time_series.begin(),
                      thrust::multiplies<hd_float>());
    stop_timer(normalise_timer);

    // Prepare the boxcar filters
    // --------------------------
    // We can't process the first and last max-filter-width/2 samples
    hd_size rel_boxcar_max = pl->params.boxcar_max/cur_dm_scrunch;
    
    hd_size max_nsamps_filtered = cur_nsamps + 1 - rel_boxcar_max;
    // This is the relative offset into the time series of the filtered data
    hd_size cur_filtered_offset = rel_boxcar_max / 2;
    // minimum filter width
    hd_size min_filter_width = std::max(cur_dm_scrunch,hd_size(1));

        
    // Create and prepare matched filtering operations
    start_timer(filter_timer);
    // Note: Filter width is relative to the current time resolution
    matched_filter_plan.prep(time_series, cur_nsamps, rel_boxcar_max);
    stop_timer(filter_timer);
    // --------------------------
    
    hd_float* filtered_series = thrust::raw_pointer_cast(&pl->d_filtered_series[0]);
    
    // Note: Filtering is done using a combination of tscrunching and
    //         'proper' boxcar convolution. The parameter min_tscrunch_width
    //         indicates how much of each to do. Raising min_tscrunch_width
    //         increases sensitivity but decreases performance and vice
    //         versa.
    
    // For each boxcar filter
    // Note: We cannot detect pulse widths < current time resolution

    // Will make it a command line option to double or linearly increase the filter width?  
    // boxcar filter loop starts 
    int boxcar_inc = pl->params.boxcar_max / pl->params.n_boxcar_inc;

    
        
      /*for( hd_size filter_width=min_filter_width;
       filter_width<=pl->params.boxcar_max;
       (if (2>1) filter_width+= boxcar_inc; 
        else filter_width*=2; ) ) {*/

      /*for( hd_size filter_width=min_filter_width;
         filter_width<=pl->params.boxcar_max;
         filter_width+= boxcar_inc) {*/

      for( hd_size filter_width=min_filter_width;
         filter_width<=pl->params.boxcar_max;
	   //         filter_width*=2 ) { // power of two boxcar
	   filter_width+=1 ) {
      hd_size rel_filter_width = filter_width / cur_dm_scrunch;
      //hd_size filter_idx = get_filter_index(filter_width); // power of two boxcar
      hd_size filter_idx = filter_width;
      
      if( pl->params.verbosity >= 4 ) {
        cout << "Filtering each beam at width of " << filter_width << endl;
      }
      
      // Note: Filter width is relative to the current time resolution
      hd_size rel_min_tscrunch_width = std::max(pl->params.min_tscrunch_width
                                                / cur_dm_scrunch,
                                                hd_size(1));
      hd_size rel_tscrunch_width = std::max(2 * rel_filter_width
                                            / rel_min_tscrunch_width,
                                            hd_size(1));
      // Filter width relative to cur_dm_scrunch AND tscrunch
      hd_size rel_rel_filter_width = rel_filter_width / rel_tscrunch_width;
      
      start_timer(filter_timer);
      
      error = matched_filter_plan.exec(filtered_series,
                                       rel_filter_width,
                                       rel_tscrunch_width);
      
      if( error != HD_NO_ERROR ) {
        return throw_error(error);
      }
      // Divide and round up
      hd_size cur_nsamps_filtered = ((max_nsamps_filtered-1)
                                     / rel_tscrunch_width + 1);
      hd_size cur_scrunch = cur_dm_scrunch * rel_tscrunch_width;
      
      // TESTING Proper normalisation
      hd_float rms = rms_getter.exec(filtered_series, cur_nsamps_filtered);
      thrust::transform(thrust::device_ptr<hd_float>(filtered_series),
                        thrust::device_ptr<hd_float>(filtered_series)
                        + cur_nsamps_filtered,
                        thrust::make_constant_iterator(hd_float(1.0)/rms),
                        thrust::device_ptr<hd_float>(filtered_series),
                        thrust::multiplies<hd_float>());

      

      hd_size prev_giant_count = d_giant_peaks.size();

      start_timer(giants_timer);


      /*if (filter_width==10 && dm_idx<500) {
	pl->d_filtered_series[1] = 7.020202;
	//if (dm_idx==150)
	//for (int idx=0;idx<cur_nsamps_filtered;idx++) cout << "VR" << pl->d_filtered_series[idx] << endl;
	}*/
      error = giant_finder.exec(filtered_series, cur_nsamps_filtered,
				pl->params.detect_thresh,
				pl->params.cand_sep_time * rel_rel_filter_width,
				d_giant_peaks,
				d_giant_inds,
				d_giant_begins,
				d_giant_ends);

      if( error != HD_NO_ERROR ) {
	return throw_error(error);
      }

      hd_size rel_cur_filtered_offset = (cur_filtered_offset /
					 rel_tscrunch_width);

      using namespace thrust::placeholders;
      thrust::transform(d_giant_inds.begin()+prev_giant_count,
			d_giant_inds.end(),
			d_giant_inds.begin()+prev_giant_count,
			/*first_idx +*/ (_1+rel_cur_filtered_offset)*cur_scrunch);
      thrust::transform(d_giant_begins.begin()+prev_giant_count,
			d_giant_begins.end(),
			d_giant_begins.begin()+prev_giant_count,
			/*first_idx +*/ (_1+rel_cur_filtered_offset)*cur_scrunch);
      thrust::transform(d_giant_ends.begin()+prev_giant_count,
			d_giant_ends.end(),
			d_giant_ends.begin()+prev_giant_count,
			/*first_idx +*/ (_1+rel_cur_filtered_offset)*cur_scrunch);

      d_giant_filter_inds.resize(d_giant_peaks.size(), filter_idx);
      d_giant_dm_inds.resize(d_giant_peaks.size(), dm_idx);
      // Note: This could be used to track total member samples if desired
      d_giant_members.resize(d_giant_peaks.size(), 1);

      stop_timer(giants_timer);
      
      // Bail if the candidate rate is too high
      hd_size total_giant_count = d_giant_peaks.size();
      hd_float data_length_mins = nsamps * pl->params.dt / 60.0;
      /*if ( pl->params.max_giant_rate && ( total_giant_count / data_length_mins > pl->params.max_giant_rate ) ) {
	too_many_giants = true;
	float searched = ((float) dm_idx * 100) / (float) dm_count;
	notrig = 1;
	cout << "WARNING: exceeded max giants/min, DM [" << dm_list[dm_idx] << "] space searched " << searched << "%" << endl;
	break;
	}*/
      
      if (total_timer.getTime() > 3.5) { // nbeams*(nsamps_gulp + max_delay + boxcar_max) * tsamp?  
	too_many_giants = true;
	float searched = ((float) dm_idx * 100) / (float) dm_count;
	cout << "WARNING: exceeded max giants processed in 3.5s, DM [" << dm_list[dm_idx] << "] space searched " << searched << "%" << endl;
	break;
      }
      
    } //close filter width loop  
    
  } //close DM loop

  hd_size giant_count = d_giant_peaks.size();
  cout << "Giant count = " << giant_count << endl;
 
  FILE *giants_out;
   char ofileg[200];
   sprintf(ofileg,"%s/giants.cand",pl->params.output_dir);
   //giants_out = fopen(ofileg,"a");
  thrust::host_vector<hd_float> h_giant_peaks;
  thrust::host_vector<hd_size>  h_giant_inds;
  thrust::host_vector<hd_size>  h_giant_begins;
  thrust::host_vector<hd_size>  h_giant_ends;
  thrust::host_vector<hd_size>  h_giant_filter_inds;
  thrust::host_vector<hd_size>  h_giant_dm_inds;
  thrust::host_vector<hd_size>  h_giant_members;
  thrust::host_vector<hd_float> h_giant_dms;

  //cout << "opened gant out file" << endl;

  h_giant_peaks = d_giant_peaks;
  h_giant_inds = d_giant_inds;
  h_giant_begins = d_giant_begins;
  h_giant_ends = d_giant_ends;
  h_giant_filter_inds = d_giant_filter_inds;
  h_giant_dm_inds = d_giant_dm_inds;
  h_giant_members = d_giant_members;

   // FILE WRITING  - prior to clustering
   hd_size samp_idx;
   hd_size beam_no;
   hd_size giant_index;
   hd_size filterbank_ind;
   hd_size block_no;
   hd_size overlap = pl->params.boxcar_max + dedisp_get_max_delay(pl->dedispersion_plan);
   hd_size block_size = nsamps - overlap;
   
   if (first_idx > 0 && gulp_idx > 1) {
   for( hd_size i=0; i<h_giant_inds.size(); ++i ) {
     if (h_giant_peaks[i] > pl->params.detect_thresh) {
     //samp_idx = first_idx + h_giant_inds[i];
     giant_index = h_giant_inds[i]%nsamps;
     beam_no = h_giant_inds[i]/nsamps + pl->params.beam;
     samp_idx = first_idx +giant_index;
         block_no = (giant_index + first_idx)/(nsamps - pl->params.boxcar_max - dedisp_get_max_delay(pl->dedispersion_plan));
         if (giant_index < overlap) filterbank_ind = block_no * block_size * pl->params.nbeams + (beam_no+1) * block_size + giant_index - overlap;
	 else filterbank_ind = block_no * block_size * pl->params.nbeams + (beam_no-1) * block_size + giant_index + nsamps - 2*overlap;

     // record output
	 // fprintf(giants_out,"print");


	 // write giants
	 /*if (giant_index < nsamps_computed + pl->params.boxcar_max/2 ) {
	   //fprintf(giants_out,"a:%g b:%lu c:%lu d:%g e:%d f:%d g:%g h:%d\n",h_giant_peaks[i],filterbank_ind, samp_idx,samp_idx * pl->params.dt,h_giant_filter_inds[i],h_giant_dm_inds[i],dm_list[h_giant_dm_inds[i]],beam_no);
	   fprintf(giants_out,"%g %lu %lu %g %d %d %g %d\n",h_giant_peaks[i],filterbank_ind, samp_idx,samp_idx * pl->params.dt,h_giant_filter_inds[i],h_giant_dm_inds[i],dm_list[h_giant_dm_inds[i]],beam_no);
     
     //cout << 'giant.cand lines test' << h_giant_peaks[i] << endl;
     }*/
     }
   }
   } 
  
  start_timer(candidates_timer);

  thrust::host_vector<hd_float> h_group_peaks;
  thrust::host_vector<hd_size>  h_group_inds;
  thrust::host_vector<hd_size>  h_group_begins;
  thrust::host_vector<hd_size>  h_group_ends;
  thrust::host_vector<hd_size>  h_group_filter_inds;
  thrust::host_vector<hd_size>  h_group_dm_inds;
  thrust::host_vector<hd_size>  h_group_members;
  thrust::host_vector<hd_float> h_group_dms;

  
  thrust::device_vector<hd_size> d_giant_labels(giant_count);
  hd_size* d_giant_labels_ptr = thrust::raw_pointer_cast(&d_giant_labels[0]);
  
  RawCandidates d_giants;
  d_giants.peaks = thrust::raw_pointer_cast(&d_giant_peaks[0]);
  d_giants.inds = thrust::raw_pointer_cast(&d_giant_inds[0]);
  d_giants.begins = thrust::raw_pointer_cast(&d_giant_begins[0]);
  d_giants.ends = thrust::raw_pointer_cast(&d_giant_ends[0]);
  d_giants.filter_inds = thrust::raw_pointer_cast(&d_giant_filter_inds[0]);
  d_giants.dm_inds = thrust::raw_pointer_cast(&d_giant_dm_inds[0]);
  d_giants.members = thrust::raw_pointer_cast(&d_giant_members[0]);
  
  hd_size filter_count = pl->params.boxcar_max;
  
  if( pl->params.verbosity >= 2 ) {
    cout << "Grouping coincident candidates..." << endl;
  }
  
  hd_size label_count;
  error = label_candidate_clusters(giant_count,
				   *(ConstRawCandidates*)&d_giants,
				   pl->params.cand_sep_time,
				   pl->params.cand_sep_filter,
				   pl->params.cand_sep_dm,
				   d_giant_labels_ptr,
				   &label_count);
  if( error != HD_NO_ERROR ) {
    return throw_error(error);
  }
  
  hd_size group_count = label_count;
  if( pl->params.verbosity >= 2 ) {
    cout << "Candidate count = " << group_count << endl;
  }

  // TESTING***
  /*std::vector<hd_byte> tdata;
  tdata.resize(100);
  for (int i=0;i<100;i++)
    tdata[i] = (hd_byte)(i);
    tfunc(tdata);*/
  

  
  thrust::device_vector<hd_float> d_group_peaks(group_count);
  thrust::device_vector<hd_size>  d_group_inds(group_count);
  thrust::device_vector<hd_size>  d_group_begins(group_count);
  thrust::device_vector<hd_size>  d_group_ends(group_count);
  thrust::device_vector<hd_size>  d_group_filter_inds(group_count);
  thrust::device_vector<hd_size>  d_group_dm_inds(group_count);
  thrust::device_vector<hd_size>  d_group_members(group_count);
  
  thrust::device_vector<hd_float> d_group_dms(group_count);
  
  RawCandidates d_groups;
  d_groups.peaks = thrust::raw_pointer_cast(&d_group_peaks[0]);
  d_groups.inds = thrust::raw_pointer_cast(&d_group_inds[0]);
  d_groups.begins = thrust::raw_pointer_cast(&d_group_begins[0]);
  d_groups.ends = thrust::raw_pointer_cast(&d_group_ends[0]);
  d_groups.filter_inds = thrust::raw_pointer_cast(&d_group_filter_inds[0]);
  d_groups.dm_inds = thrust::raw_pointer_cast(&d_group_dm_inds[0]);
  d_groups.members = thrust::raw_pointer_cast(&d_group_members[0]);
  
  merge_candidates(giant_count,
		   d_giant_labels_ptr,
		   *(ConstRawCandidates*)&d_giants,
		   d_groups);
  
  // Look up the actual DM of each group
  thrust::device_vector<hd_float> d_dm_list(dm_list, dm_list+dm_count);
  thrust::gather(d_group_dm_inds.begin(), d_group_dm_inds.end(),
		 d_dm_list.begin(),
		 d_group_dms.begin());
  
  // Device to host transfer of candidates
  h_group_peaks = d_group_peaks;
  h_group_inds = d_group_inds;
  h_group_begins = d_group_begins;
  h_group_ends = d_group_ends;
  h_group_filter_inds = d_group_filter_inds;
  h_group_dm_inds = d_group_dm_inds;
  h_group_members = d_group_members;
  h_group_dms = d_group_dms;




  
  // writing stuff
  char buffer[64];
  time_t now = pl->params.utc_start + (time_t) (first_idx / pl->params.spectra_per_second);
  strftime (buffer, 64, HD_TIMESTR, (struct tm*) gmtime(&now));

  std::stringstream ss;
  ss << std::setw(2) << std::setfill('0') << (pl->params.beam)%13+1;

  std::ostringstream oss;

  if ( pl->params.coincidencer_host != NULL && pl->params.coincidencer_port != -1 )
  {
    try 
    {
      ClientSocket client_socket ( pl->params.coincidencer_host, pl->params.coincidencer_port );

      strftime (buffer, 64, HD_TIMESTR, (struct tm*) gmtime(&(pl->params.utc_start)));

      //oss <<  buffer << " ";

      time_t now = pl->params.utc_start + (time_t) (first_idx / pl->params.spectra_per_second);
      strftime (buffer, 64, HD_TIMESTR, (struct tm*) gmtime(&now));
      //oss << buffer << " ";

      /*oss << first_idx << endl;
      //oss << ss.str() << " ";
      //oss << h_group_peaks.size() << endl;
      client_socket << oss.str();
      oss.flush();
      oss.str("");

      for (hd_size i=0; i<h_group_peaks.size(); ++i ) 
      {
        hd_size samp_idx = first_idx + h_group_inds[i];
        oss << h_group_peaks[i] << "\t"
                      << samp_idx << "\t"
                      << samp_idx * pl->params.dt << "\t"
                      << h_group_filter_inds[i] << "\t"
                      << h_group_dm_inds[i] << "\t"
                      << h_group_dms[i] << "\t"
                      << h_group_members[i] << "\t"
                      << first_idx + h_group_begins[i] << "\t"
                      << first_idx + h_group_ends[i] << endl;

        client_socket << oss.str();
        oss.flush();
        oss.str("");
	}*/

      // gc: write giants in socket 
      oss << gulp_idx << endl;
      //oss << ss.str() << " ";
      //oss << h_giant_inds.size() << endl;
      client_socket << oss.str();
      oss.flush();
      oss.str("");

      if (gulp_idx > 1 && giant_count < 10000) {

	for( hd_size i=0; i<h_giant_inds.size(); ++i ) {
	  if (h_giant_peaks[i] > pl->params.detect_thresh) {
	    //samp_idx = first_idx + h_giant_inds[i];
	    giant_index = h_giant_inds[i]%nsamps;
	    beam_no = h_giant_inds[i]/nsamps + pl->params.beam;
	    samp_idx = first_idx +giant_index;
	    block_no = (giant_index + first_idx)/(nsamps - pl->params.boxcar_max - dedisp_get_max_delay(pl->dedispersion_plan));
	    if (giant_index < overlap) filterbank_ind = block_no * block_size * pl->params.nbeams + (beam_no+1) * block_size + giant_index - overlap;
	    else filterbank_ind = block_no * block_size * pl->params.nbeams + (beam_no-1) * block_size + giant_index + nsamps - 2*overlap;
	    // record output  
	    if (giant_index < nsamps_computed + pl->params.boxcar_max/2) {
	      oss << h_giant_peaks[i] << " "
		  << filterbank_ind << " "
		  << samp_idx << " " 
		  << samp_idx * pl->params.dt << " "
		  << h_giant_filter_inds[i] << " "
		  << h_giant_dm_inds[i] << " "
		  << dm_list[h_giant_dm_inds[i]] << " "
		  << beam_no << endl;
	      
	      client_socket << oss.str();
	      oss.flush();
	      oss.str("");
	    }
	  }
	}
	// client_socket should close when it goes out of scope...
      }
    }
    catch (SocketException& e )
      {
	std::cerr << "SocketException was caught:" << e.description() << "\n";
      }
      
    
  }
    
   FILE *cands_out;
   char ofile[200];
   float S1, S2;
   sprintf(ofile,"%s/heimdall.cand",pl->params.output_dir);
   cands_out = fopen(ofile,"a");

   
   // FILE WRITING VR
   float dm, snr;
   char cmd[300];
   hd_size rawsample;
   int samp, wid;
   char filname[200];
   int s1, s2;
   
   int maxI=-1;
   float maxSNR=0.;
   float maxFRB=0.;
   
   std::vector<hd_byte> output_data;
   int sent=0;
   hd_size samp_idx2;
   hd_size group_beam_no;   
   hd_size group_sample_ind;
   hd_size block_no2;
   hd_size filterbank_ind2;
   for( hd_size i=0; i<h_group_peaks.size(); ++i ) {

     group_sample_ind = h_group_inds[i]%nsamps;
     group_beam_no = h_group_inds[i]/nsamps + pl->params.beam;
     samp_idx2 = first_idx + group_sample_ind; 
     block_no2 = (group_sample_ind + first_idx)/(nsamps - pl->params.boxcar_max - dedisp_get_max_delay(pl->dedispersion_plan));
     if (group_sample_ind < overlap) filterbank_ind2 = block_no2 * block_size * pl->params.nbeams + (beam_no-1) * block_size + group_sample_ind - overlap;
     else filterbank_ind2 = block_no2 * block_size * pl->params.nbeams + (beam_no-1) * block_size + group_sample_ind + nsamps - 2*overlap;
     // record output
     if (group_sample_ind < *nsamps_processed && gulp_idx > 1) fprintf(cands_out,"%g %lu %lu %g %d %d %g %d %d\n",h_group_peaks[i],filterbank_ind2,samp_idx2,samp_idx2 * pl->params.dt,h_group_filter_inds[i],h_group_dm_inds[i],h_group_dms[i],h_group_members[i],group_beam_no);
     
     // if pulse is dump-able
     if (((h_group_peaks[i]>8.0 && group_sample_ind < nsamps_computed && h_group_filter_inds[i]<14 && h_group_dms[i]>20.0) || (h_group_peaks[i]>7.0 && h_group_dms[i]>52.0 && h_group_dms[i]<63.0 && group_sample_ind < nsamps_computed && h_group_filter_inds[i]<3)) && (gulp_idx > 1)) {

       // find peak SNR so we're only dumping one per block
       if (h_group_peaks[i]>maxSNR) {
	 maxSNR = h_group_peaks[i];
	 maxI = i;
       }
       
     }
     
   }

   // check for too many cands per block. 
   if (h_group_peaks.size()>0 && maxI!=-1) {

     samp_idx = first_idx + h_group_begins[maxI];
     s1 = h_group_begins[maxI]-50;
     if (s1<0) s1=0;
     s2 = h_group_ends[maxI]+int((0.000761*h_group_dms[maxI])/pl->params.dt)+50+(int)(h_group_filter_inds[maxI]);
     if (s2>nbytes/(pl->params.nchans*nbits/8)) s2=nbytes/(pl->params.nchans*nbits/8);
	 
     output_data.resize((s2-s1)*(pl->params.nchans*nbits/8));
     std::copy(pl->h_clean_filterbank.begin()+s1*(pl->params.nchans*nbits/8),pl->h_clean_filterbank.begin()+s2*(pl->params.nchans*nbits/8),output_data.begin());
     
     sprintf(filname,"%s/candidate_%g_%g_%d_%d_%d.fil",pl->params.output_dir,h_group_peaks[maxI],h_group_dms[maxI],h_group_filter_inds[maxI],first_idx + (h_group_inds[maxI]%nsamps),h_group_inds[maxI]/nsamps + pl->params.beam);
     output = fopen(filname,"wb");
     send_string("HEADER_START");
     send_string("source_name");
     send_string("DSATEST");
     send_int("machine_id",1);
     send_int("telescope_id",82);
     send_int("data_type",1); // filterbank data
     send_double("fch1",pl->params.f0);
     send_double("foff",pl->params.df);
     send_int("nchans",pl->params.nchans);
     send_int("nbits",nbits);
     send_double("tstart",55000.0);
     send_double("tsamp",pl->params.dt);
     send_int("nifs",1);
     send_string("HEADER_END");
     
     fwrite((&output_data[0]),nbits/8,pl->params.nchans*(s2-s1),output);
     fclose(output);
            
   }
   
   //fclose(giants_out);     
   fclose(cands_out);
   stop_timer(candidates_timer);


   stop_timer(total_timer);
   
   cout << "Mem alloc time:          " << memory_timer.getTime() << endl;
   cout << "0-DM cleaning time:      " << clean_timer.getTime() << endl;
   cout << "Dedispersion time:       " << dedisp_timer.getTime() << endl;
   cout << "Copy time:               " << copy_timer.getTime() << endl;
   cout << "Baselining time:         " << baseline_timer.getTime() << endl;
   cout << "Normalisation time:      " << normalise_timer.getTime() << endl;
   cout << "Filtering time:          " << filter_timer.getTime() << endl;
   cout << "Find giants time:        " << giants_timer.getTime() << endl;
   cout << "Process candidates time: " << candidates_timer.getTime() << endl;
   cout << "Total time:              " << total_timer.getTime() << endl;
   
  if( too_many_giants ) {
    return HD_TOO_MANY_EVENTS;
  }
  else {
    return HD_NO_ERROR;
  }
  
  
  
}

void hd_destroy_pipeline(hd_pipeline pipeline) {
  if( pipeline->params.verbosity >= 2 ) {
    cout << "\tDeleting pipeline object..." << endl;
  }
  
  dedisp_destroy_plan(pipeline->dedispersion_plan);
  
  // Note: This assumes memory owned by pipeline cleans itself up
  if( pipeline ) {
    delete pipeline;
  }
}
